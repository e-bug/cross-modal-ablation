#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

template<typename x_t, typename y_t, typename out_t>
struct AxpbyFunctor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<3>& tl,
    float a,
    float b,
    int arg_to_check)
  {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    x_t* x = (x_t*)tl.addresses[0][tensor_loc];
    x += chunk_idx*chunk_size;

    y_t* y = (y_t*)tl.addresses[1][tensor_loc];
    y += chunk_idx*chunk_size;

    out_t* out = (out_t*)tl.addresses[2][tensor_loc];
    out += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    // Non-divergent exit condition for __syncthreads, not necessary here
    float xs[ILP];
    float ys[ILP];
    for(int i_start = 0;
        i_start < n && i_start < chunk_size;
        i_start += blockDim.x*ILP)
    {
      #pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        xs[ii] = 0;
        ys[ii] = 0;
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          xs[ii] = static_cast<float>(x[i]);
          ys[ii] = static_cast<float>(y[i]);
        }
      }

      // see note in multi_tensor_scale_kernel.cu
      #pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          out[i] = static_cast<out_t>(a*xs[ii] + b*ys[ii]);
          bool finite = true;
          if(arg_to_check == -1)
            finite = (isfinite(xs[ii]) && isfinite(ys[ii]));
          if(arg_to_check == 0)
            finite = isfinite(xs[ii]);
          if(arg_to_check == 1)
            finite = isfinite(ys[ii]);
          if(!finite)
            *noop_gmem = 1; // Blindly fire off a write.  These will race but that's ok.
        }
      }
    }
  }
};

void multi_tensor_axpby_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  float a,
  float b,
  int arg_to_check)
{
  using namespace at;
  // The output (downscaled) type is always float.
  // If build times suffer, think about where to put this dispatch,
  // and what logic should be moved out of multi_tensor_apply.

  DISPATCH_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 0, "multi_tensor_axpby_cuda",
    DISPATCH_FLOAT_AND_HALF(tensor_lists[1][0].scalar_type(), 1, "multi_tensor_axpby_cuda",
      DISPATCH_FLOAT_AND_HALF(tensor_lists[2][0].scalar_type(), 2, "multi_tensor_axpby_cuda",
           multi_tensor_apply<3>(
             BLOCK_SIZE,
             chunk_size,
             noop_flag,
             tensor_lists,
             AxpbyFunctor<scalar_t_0, scalar_t_1, scalar_t_2>(),
             a,
             b,
             arg_to_check); )))

  AT_CUDA_CHECK(hipGetLastError());

  // AT_CUDA_CHECK(hipDeviceSynchronize());
}
